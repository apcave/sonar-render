#include "hip/hip_runtime.h"
#include "CudaModelTes.cuh"
#include "GeoMath.h"
#include "dcomplex.h"
#include <stdio.h>

using namespace std;

int CudaModelTes::SetGlobalParameters(dcomplex k_wave, float pixel_delta)
{
    printf("SetGlobalParameters .......\n");
    // Set the global parameters for the GPU
    hipMalloc(&dev_k_wave, 1 * sizeof(dcomplex));
    hipError_t cudaStatus = hipMemcpy(dev_k_wave, &k_wave, sizeof(dcomplex), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "k_wave failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    printf("pixel_delta: %f\n", pixel_delta);
    hipMalloc(&dev_pixel_delta, 1 * sizeof(float));
    cudaStatus = hipMemcpy(dev_pixel_delta, &pixel_delta, sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "pixel_delta failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    return 0;
}

int CudaModelTes::MakeSourcePointsOnGPU(vector<PressurePoint *> source_points)
{
    host_num_source_points = source_points.size();
    printf("MakeSourcePointsOnGPU (num pnts %d) .......\n", host_num_source_points);

    auto position = new float3[host_num_source_points];
    auto pressure = new dcomplex[host_num_source_points];
    for (int i = 0; i < host_num_source_points; i++)
    {
        position[i] = source_points[i]->position;
        pressure[i] = source_points[i]->pressure;
    }
    // Allocate memory for the source points on the device
    hipMalloc(&dev_source_points_position, host_num_source_points * sizeof(float3));
    hipMemcpy(dev_source_points_position, position, host_num_source_points * sizeof(float3), hipMemcpyHostToDevice);
    hipMalloc(&dev_source_points_pressure, host_num_source_points * sizeof(dcomplex));
    hipMemcpy(dev_source_points_pressure, pressure, host_num_source_points * sizeof(dcomplex), hipMemcpyHostToDevice);
    // Free the host memory
    delete[] position;
    delete[] pressure;
    return 0;
}

int CudaModelTes::MakeFieldPointsOnGPU(vector<PressurePoint *> field_points)
{

    host_num_field_points = field_points.size();

    auto position = new float3[host_num_field_points];
    auto pressure = new dcomplex[host_num_field_points];
    for (int i = 0; i < host_num_field_points; i++)
    {
        position[i] = field_points[i]->position;
        pressure[i] = field_points[i]->pressure;
    }
    // Allocate memory for the source points on the device
    hipMalloc(&dev_field_points_position, host_num_field_points * sizeof(float3));
    hipMemcpy(dev_field_points_position, position, host_num_field_points * sizeof(float3), hipMemcpyHostToDevice);
    hipMalloc(&dev_field_points_pressure, host_num_field_points * sizeof(dcomplex));
    hipMemcpy(dev_field_points_pressure, pressure, host_num_field_points * sizeof(dcomplex), hipMemcpyHostToDevice);
    // Free the host memory
    delete[] position;
    delete[] pressure;

    return 0;
}

int CudaModelTes::MakeObjectOnGPU(vector<Facet *> facets)
{
    // Copy the facet data to the GPU
    printf("MakeObjectOnGPU .......\n");

    int number_of_facets = facets.size();
    host_object_num_facets.push_back(number_of_facets);

    float **dev_Facets_PixelArea;
    hipMalloc(&dev_Facets_PixelArea, number_of_facets * sizeof(float *));
    dev_Object_Facets_PixelArea.push_back(dev_Facets_PixelArea);

    float **host_PixelArea = new float *[number_of_facets];
    for (int i = 0; i < number_of_facets; i++)
    {
        int ArrLen = facets[i]->NumXpnts * facets[i]->NumYpnts;
        // Allocate the Area for the pixel array on the host.
        hipMalloc(&host_PixelArea[i], ArrLen * sizeof(float));
        // Copy the pixel area data to host CUDA to the device.
        hipMemcpy(host_PixelArea[i], facets[i]->PixelArea, ArrLen * sizeof(float), hipMemcpyHostToDevice);
    }
    // Copy the pixel area data to the device.
    hipMemcpy(dev_Facets_PixelArea, host_PixelArea, number_of_facets * sizeof(float *), hipMemcpyHostToDevice);

    dcomplex **dev_Facets_Pressure;
    hipMalloc(&dev_Facets_Pressure, number_of_facets * sizeof(dcomplex *));
    dev_Object_Facets_Pressure.push_back(dev_Facets_Pressure);
    dcomplex **host_Pressure = new dcomplex *[number_of_facets];
    for (int i = 0; i < number_of_facets; i++)
    {
        int ArrLen = facets[i]->NumXpnts * facets[i]->NumYpnts;
        // Allocate the Area for the pixel pressure array on the host.
        hipMalloc(&host_Pressure[i], ArrLen * sizeof(dcomplex));
        // Copy the pixel pressure data to host CUDA to the device.
        hipMemcpy(host_Pressure[i], facets[i]->PressureValues, ArrLen * sizeof(dcomplex), hipMemcpyHostToDevice);
    }
    // Copy the pixel pressure data to the device.
    hipMemcpy(dev_Facets_Pressure, host_Pressure, number_of_facets * sizeof(dcomplex *), hipMemcpyHostToDevice);

    int facet_cnt = 0;
    auto host_Facets_points = new int3[number_of_facets];
    auto host_Facets_Normals = new float3[number_of_facets];
    auto host_base_points = new float3[number_of_facets];
    auto host_Facets_xAxis = new float3[number_of_facets];
    auto host_Facets_yAxis = new float3[number_of_facets];
    for (int i = 0; i < number_of_facets; i++)
    {
        // Memory for dev_Facets_PixelArea[facet_cnt] is already allocated in the initialization loop above.
        host_Facets_points[i].x = facets[i]->NumXpnts;
        host_Facets_points[i].y = facets[i]->NumYpnts;
        host_Facets_points[i].z = facets[i]->NumXpntsNegative;
        host_Facets_Normals[i] = facets[i]->Normal;
        host_base_points[i] = facets[i]->pointOnBase;
        host_Facets_xAxis[i] = facets[i]->xAxis;
        host_Facets_yAxis[i] = facets[i]->yAxis;
        facet_cnt++;
    }

    int3 *dev_Facets_points;
    hipMalloc(&dev_Facets_points, number_of_facets * sizeof(int3));
    hipMemcpy(dev_Facets_points, host_Facets_points, number_of_facets * sizeof(int3), hipMemcpyHostToDevice);
    dev_Object_Facets_points.push_back(dev_Facets_points);
    host_Object_Facets_points.push_back(host_Facets_points);

    float3 *dev_Facets_Normals;
    hipMalloc(&dev_Facets_Normals, number_of_facets * sizeof(float3));
    hipMemcpy(dev_Facets_Normals, host_Facets_Normals, number_of_facets * sizeof(float3), hipMemcpyHostToDevice);
    dev_Object_Facets_Normals.push_back(dev_Facets_Normals);

    float3 *dev_base_points;
    hipMalloc(&dev_base_points, number_of_facets * sizeof(float3));
    hipMemcpy(dev_base_points, host_base_points, number_of_facets * sizeof(float3), hipMemcpyHostToDevice);
    dev_Object_base_points.push_back(dev_base_points);

    float3 *dev_Facets_xAxis;
    hipMalloc(&dev_Facets_xAxis, number_of_facets * sizeof(float3));
    hipMemcpy(dev_Facets_xAxis, host_Facets_xAxis, number_of_facets * sizeof(float3), hipMemcpyHostToDevice);
    dev_Object_Facets_xAxis.push_back(dev_Facets_xAxis);

    float3 *dev_Facets_yAxis;
    hipMalloc(&dev_Facets_yAxis, number_of_facets * sizeof(float3));
    hipMemcpy(dev_Facets_yAxis, host_Facets_yAxis, number_of_facets * sizeof(float3), hipMemcpyHostToDevice);
    dev_Object_Facets_yAxis.push_back(dev_Facets_yAxis);

    delete[] host_PixelArea;
    delete[] host_Pressure;
    delete[] host_Facets_Normals;
    delete[] host_base_points;

    printf("Allocated object memory on GPU.\n");
    return 0;
}

int CudaModelTes::StartCuda()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        return 1;
    }
    printf("CUDA started successfully.\n");
    return 0;
}

int CudaModelTes::StopCuda()
{
    // hipError_t cudaStatus = hipUnbindTexture(dev_Positions);
    // if (cudaStatus != hipSuccess)
    // {
    //     printf("Unbinding of Positions Texture Failed!\n");
    //     return 1;
    // }

    // cudaStatus = hipDeviceReset();
    // if (cudaStatus != hipSuccess)
    // {
    //     fprintf(stderr, "hipDeviceReset failed!\n");
    //     return 1;
    // }
    return 0;
}

__global__ void childKernel(int *data)
{
    // int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("ThreadIdx.x: %d, ThreadIdx.y: %d, blockIdx.x: %d, blockDim.x: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x);
    // printf("Child kernel: Thread %d, data[%d] = %d\n", idx, idx, data[idx]);
}

int CudaModelTes::TestGPU()
{
    int *dev_data;
    int host_data[10] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};

    // Allocate memory on the device
    hipError_t cudaStatus = hipMalloc(&dev_data, 10 * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "test malloc failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    cudaStatus = hipMemcpy(dev_data, host_data, 10 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "test memory copy failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    int h_num_xpnts = 10; // Example value
    int h_num_ypnts = 10; // Example value
    dim3 threadsPerBlock(10, 10);
    dim3 numBlocks((h_num_xpnts + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (h_num_ypnts + threadsPerBlock.y - 1) / threadsPerBlock.y);
    printf("numBlocks.x: %d, numBlocks.y: %d\n", numBlocks.x, numBlocks.y);
    childKernel<<<numBlocks, threadsPerBlock>>>(dev_data);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipDeviceSynchronize();
    return 0;
}

int CudaModelTes::DoCalculations()
{
    printf("DoCalculations .......\n");
    // TestGPU();
    ProjectSourcePointsToFacet();
    ProjectFromFacetsToFieldPoints();
    return 0;
}

__device__ double atomicAddDouble(double *address, double val)
{
    unsigned long long int *address_as_ull = (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(__longlong_as_double(assumed) + val));
    } while (assumed != old);

    return __longlong_as_double(old);
}

/**
 * @brief Kernel to waves from a point source to a facet.
 *
 * Note all pointers are to memory addressses on the device.
 * This function is not to be used with matrix compression.
 * The calculations are done in the global coordinate system.
 */
__global__ void ProjectSourcePointToFacetKernel(
    dcomplex *k_wave,
    float *pixel_delta,
    int source_point_num,
    int facet_num,
    float3 *source_points_position,
    dcomplex *source_points_pressure,
    int3 *facet_Points,
    float3 *base_points,
    float3 *facets_xaxis,
    float3 *facets_yaxis,
    float **facets_PixelArea,
    dcomplex **facets_Pressure)
{
    dcomplex k = *k_wave;
    float delta = *pixel_delta;

    // printf("k_wave: %f, %f\n", k.r, k.i);
    // printf("pixel_delta: %f\n", delta);

    // Kernel code to project point to point
    // printf("ThreadIdx.x: %d, ThreadIdx.y: %d, blockIdx.x: %d, blockDim.x: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x);
    int xPnt = threadIdx.x;
    int yPnt = threadIdx.y;

    int NumXpnts = facet_Points[facet_num].x;
    // int NumYpnts = facet_Points[facet_num].y;
    int NumXpntsNegative = facet_Points[facet_num].z;

    if (facets_PixelArea[facet_num][yPnt * NumXpnts + xPnt] == 0)
    {
        // printf("facets_PixelArea is zero\n");
        return;
    }

    // printf("xPnt: %d, yPnt: %d\n", xPnt, yPnt);
    // printf("NumXpnts: %d, NumYpnts: %d, NumXpntsNegative: %d\n", NumXpnts, NumYpnts, NumXpntsNegative);

    float3 P1g = source_points_position[source_point_num];
    dcomplex source_pressure = source_points_pressure[source_point_num];

    // printf("Source Point: %f, %f, %f\n", P1g.x, P1g.y, P1g.z);
    // printf("Source Pressure: %f, %f\n", source_pressure.r, source_pressure.i);

    // This is the x offset from the base point to the approximate centriod of the pixel.
    float xoffset = delta * (xPnt - NumXpntsNegative) + delta / 2; // This value can be negative.
    // This is the y offset from the base point to the approximate centriod of the pixel.
    float yoffset = delta * yPnt + delta / 2;

    float3 xAxis = facets_xaxis[facet_num];
    float3 yAxis = facets_yaxis[facet_num];

    xAxis.x = xoffset * xAxis.x;
    xAxis.y = xoffset * xAxis.y;
    xAxis.z = xoffset * xAxis.z;

    yAxis.x = yoffset * yAxis.x;
    yAxis.y = yoffset * yAxis.y;
    yAxis.z = yoffset * yAxis.z;

    float3 facet_base = base_points[facet_num];
    float3 P2g;
    P2g.x = xAxis.x + yAxis.x + facet_base.x;
    P2g.y = xAxis.y + yAxis.y + facet_base.y;
    P2g.z = xAxis.z + yAxis.z + facet_base.z;
    // printf("Facet Point Global Ref: %f, %f, %f\n", P2g.x, P2g.y, P2g.z);

    // The distance from the source point to the facet point.
    float r_sf = sqrtf((P1g.x - P2g.x) * (P1g.x - P2g.x) + (P1g.y - P2g.y) * (P1g.y - P2g.y) + (P1g.z - P2g.z) * (P1g.z - P2g.z));

    // printf("Distance from source to facet: %f\n", r_sf);

    // P2 = P2*exp(-i*k*r_sf)
    dcomplex i = devComplex(0, 1);
    dcomplex var = devCmul(i, k);
    var = devRCmul(r_sf, var);
    var = devCexp(var);                  // This has phase and attenuation.
    var = devCmul(var, source_pressure); // This includes the orginal pressure.
    // printf("Pressure prior to spreading at facet point: %f, %f\n", var.r, var.i);

    // Area1 = Pressure the 1Pa over 1m^2
    // Area2 = 4 * PI * r_sf * r_sf
    // atten_spread = Area1 / Area2 <--- important for other projections.
    float att_spread = 1 / (4 * M_PI * r_sf * r_sf);
    var = devRCmul(att_spread, var);
    // printf("Spherical spread: %f\n", att_spread);

    if (devCabs(var) > 1.0)
    {
        printf("Pressure is too large to add to facet point.\n");
        printf("Spherical spread: %e\n", att_spread);
        printf("Pressure add to field point prior to spreading: %e, %e\n", var.r, var.i);
        return;
    }

    // printf("Pressure at facet point: %f, %f\n", var.r, var.i);

    // Save the pressure to the facet pressure array.
    // Note var may be small and accumulate over may projects that why the complex numbers are doubles.
    atomicAddDouble(&(facets_Pressure[facet_num][yPnt * NumXpnts + xPnt].r), var.r);
    atomicAddDouble(&(facets_Pressure[facet_num][yPnt * NumXpnts + xPnt].i), var.i);
}

int CudaModelTes::ProjectSourcePointsToFacet()
{
    // Every facet can have a different number of points.
    printf("Host ProjectPointToFacet....\n");

    for (int source_point_num = 0; source_point_num < host_num_source_points; source_point_num++)
    {
        for (int object_num = 0; object_num < host_object_num_facets.size(); object_num++)
        {
            for (int facet_num = 0; facet_num < host_object_num_facets[object_num]; facet_num++)
            {
                int3 h_Facets_points = host_Object_Facets_points[object_num][facet_num];

                dim3 threadsPerBlock(h_Facets_points.x, h_Facets_points.y);
                dim3 numBlocks(1, 1);

                // printf("ThreadsPerBlock.x: %d, threadsPerBlock.y: %d\n", threadsPerBlock.x, threadsPerBlock.y);
                // printf("numBlocks.x: %d, numBlocks.y: %d\n", numBlocks.x, numBlocks.y);

                ProjectSourcePointToFacetKernel<<<numBlocks, threadsPerBlock>>>(
                    dev_k_wave,
                    dev_pixel_delta,
                    source_point_num,
                    facet_num,
                    dev_source_points_position,
                    dev_source_points_pressure,
                    dev_Object_Facets_points[object_num],
                    dev_Object_base_points[object_num],
                    dev_Object_Facets_xAxis[object_num],
                    dev_Object_Facets_yAxis[object_num],
                    dev_Object_Facets_PixelArea[object_num],
                    dev_Object_Facets_Pressure[object_num]);

                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                {
                    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
                    return 1;
                }
            }
        }
    }

    // More testing is required on large models to see how CUDA manages the cores.
    hipDeviceSynchronize();
    return 0;
}

__global__ void ProjectSourceFacetToFeildPointKernel(
    dcomplex *k_wave,
    float *pixel_delta,
    int field_point_num,
    int facet_num,
    float3 *field_points_position,
    dcomplex *field_points_pressure,
    int3 *facet_Points,
    float3 *base_points,
    float3 *facets_xaxis,
    float3 *facets_yaxis,
    float **facets_PixelArea,
    dcomplex **facets_Pressure)
{
    dcomplex k = *k_wave;
    float delta = *pixel_delta;

    // printf("k_wave: %f, %f\n", k.r, k.i);
    // printf("pixel_delta: %f\n", delta);

    // Kernel code to project point to point
    // printf("ThreadIdx.x: %d, ThreadIdx.y: %d, blockIdx.x: %d, blockDim.x: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x);
    int xPnt = threadIdx.x;
    int yPnt = threadIdx.y;

    int NumXpnts = facet_Points[facet_num].x;
    // int NumYpnts = facet_Points[facet_num].y;
    int NumXpntsNegative = facet_Points[facet_num].z;

    float pixel_area = facets_PixelArea[facet_num][yPnt * NumXpnts + xPnt];
    if (pixel_area == 0)
    {
        // printf("facets_PixelArea is zero\n");
        return;
    }

    // printf("xPnt: %d, yPnt: %d\n", xPnt, yPnt);
    // printf("NumXpnts: %d, NumYpnts: %d, NumXpntsNegative: %d\n", NumXpnts, NumYpnts, NumXpntsNegative);

    float3 P2g = field_points_position[field_point_num];
    // printf("Field Point: %f, %f, %f\n", P2g.x, P2g.y, P2g.z);

    // This is the x offset from the base point to the approximate centriod of the pixel.
    float xoffset = delta * (xPnt - NumXpntsNegative) + delta / 2; // This value can be negative.
    // This is the y offset from the base point to the approximate centriod of the pixel.
    float yoffset = delta * yPnt + delta / 2;

    float3 xAxis = facets_xaxis[facet_num];
    float3 yAxis = facets_yaxis[facet_num];

    xAxis.x = xoffset * xAxis.x;
    xAxis.y = xoffset * xAxis.y;
    xAxis.z = xoffset * xAxis.z;

    yAxis.x = yoffset * yAxis.x;
    yAxis.y = yoffset * yAxis.y;
    yAxis.z = yoffset * yAxis.z;

    float3 facet_base = base_points[facet_num];
    float3 P1g;
    P1g.x = xAxis.x + yAxis.x + facet_base.x;
    P1g.y = xAxis.y + yAxis.y + facet_base.y;
    P1g.z = xAxis.z + yAxis.z + facet_base.z;
    // printf("Facet Point Global Ref: %f, %f, %f\n", P1g.x, P1g.y, P1g.z);

    dcomplex source_pressure = facets_Pressure[facet_num][yPnt * NumXpnts + xPnt];

    // The distance from the source point to the facet point.
    float r_sf = sqrtf((P1g.x - P2g.x) * (P1g.x - P2g.x) + (P1g.y - P2g.y) * (P1g.y - P2g.y) + (P1g.z - P2g.z) * (P1g.z - P2g.z));

    // printf("Distance from pixel to field point: %f\n", r_sf);

    // P2 = P2*exp(-i*k*r_sf)
    dcomplex i = devComplex(0, 1);
    dcomplex var = devCmul(i, k);
    var = devRCmul(r_sf, var);
    var = devCexp(var);                  // This has phase and attenuation.
    var = devCmul(var, source_pressure); // This includes the orginal pressure.
    // printf("Pressure add to field point prior to spreading: %f, %f\n", var.r, var.i);

    // Area1 = Pressure the 1Pa over 1m^2
    // Area2 = 4 * PI * r_sf * r_sf
    // atten_spread = Area1 / Area2 <--- important for other projections.
    float att_spread = pixel_area / (4 * M_PI * r_sf * r_sf);
    var = devRCmul(att_spread, var);
    // printf("Spherical spread: %f\n", att_spread);

    if (devCabs(var) > 1.0)
    {
        printf("Pressure is too large to add to field point.\n");
        printf("r_sf: %f\n", r_sf);
        printf("source_pressure: %e, %e\n", source_pressure.r, source_pressure.i);
        printf("Spherical spread: %e\n", att_spread);
        printf("Pressure add to field point prior to spreading: %e, %e\n", var.r, var.i);
        return;
    }

    // printf("Pressure added to field point: %f, %f\n", var.r, var.i);

    // Save the pressure to the facet pressure array.
    // Note var may be small and accumulate over may projects that why the complex numbers are doubles.
    atomicAddDouble(&(field_points_pressure[field_point_num].r), var.r);
    atomicAddDouble(&(field_points_pressure[field_point_num].i), var.i);
}

int CudaModelTes::ProjectFromFacetsToFieldPoints()
{
    printf("ProjectFromFacetsToFieldPoints .......\n");

    for (int object_num = 0; object_num < host_object_num_facets.size(); object_num++)
    {

        for (int facet_num = 0; facet_num < host_object_num_facets[object_num]; facet_num++)
        {

            for (int field_point_num = 0; field_point_num < host_num_field_points; field_point_num++)
            {

                int3 h_Facets_points = host_Object_Facets_points[object_num][facet_num];

                dim3 threadsPerBlock(h_Facets_points.x, h_Facets_points.y);
                dim3 numBlocks(1, 1);

                // printf("ThreadsPerBlock.x: %d, threadsPerBlock.y: %d\n", threadsPerBlock.x, threadsPerBlock.y);
                // printf("numBlocks.x: %d, numBlocks.y: %d\n", numBlocks.x, numBlocks.y);

                ProjectSourceFacetToFeildPointKernel<<<numBlocks, threadsPerBlock>>>(
                    dev_k_wave,
                    dev_pixel_delta,
                    field_point_num,
                    facet_num,
                    dev_field_points_position,
                    dev_field_points_pressure,
                    dev_Object_Facets_points[object_num],
                    dev_Object_base_points[object_num],
                    dev_Object_Facets_xAxis[object_num],
                    dev_Object_Facets_yAxis[object_num],
                    dev_Object_Facets_PixelArea[object_num],
                    dev_Object_Facets_Pressure[object_num]);

                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                {
                    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
                    return 1;
                }
            }
        }
    }
    // More testing is required on large models to see how CUDA manages the cores.
    hipDeviceSynchronize();
    return 0;
}

int CudaModelTes::GetFieldPointValGPU(dcomplex *field_points_pressure)
{
    // Copy the field point pressures from the device to the host.
    hipMemcpy(field_points_pressure, dev_field_points_pressure, host_num_field_points * sizeof(dcomplex), hipMemcpyDeviceToHost);
    return 0;
}