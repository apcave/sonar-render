#include "hip/hip_runtime.h"
#include "CudaModelTes.cuh"
#include "GeoMath.h"
#include "dcomplex.h"
#include "CudaUtils.cuh"

#include <stdio.h>

/**
 * @brief Kernel to waves from a point source to a facet.
 *
 * Note all pointers are to memory addressses on the device.
 * This function is not to be used with matrix compression.
 * The calculations are done in the global coordinate system.
 */
__global__ void ProjectSourcePointToFacetKernel(
    dcomplex *k_wave,
    float *pixel_delta,
    int source_point_num,
    int facet_num,
    float3 *source_points_position,
    dcomplex *source_points_pressure,
    int3 *facet_Points,
    float3 *base_points,
    float3 *facets_xaxis,
    float3 *facets_yaxis,
    float **facets_PixelArea,
    dcomplex **facets_Pressure)
{
    dcomplex k = *k_wave;
    float delta = *pixel_delta;

    // printf("k_wave: %f, %f\n", k.r, k.i);
    // printf("pixel_delta: %f\n", delta);

    // Kernel code to project point to point
    // printf("ThreadIdx.x: %d, ThreadIdx.y: %d, blockIdx.x: %d, blockDim.x: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x);
    int xPnt = threadIdx.x;
    int yPnt = threadIdx.y;

    int NumXpnts = facet_Points[facet_num].x;
    // int NumYpnts = facet_Points[facet_num].y;
    int NumXpntsNegative = facet_Points[facet_num].z;

    if (facets_PixelArea[facet_num][yPnt * NumXpnts + xPnt] == 0)
    {
        // printf("facets_PixelArea is zero\n");
        return;
    }

    // printf("xPnt: %d, yPnt: %d\n", xPnt, yPnt);
    // printf("NumXpnts: %d, NumYpnts: %d, NumXpntsNegative: %d\n", NumXpnts, NumYpnts, NumXpntsNegative);

    float3 P1g = source_points_position[source_point_num];
    dcomplex source_pressure = source_points_pressure[source_point_num];

    // printf("Source Point: %f, %f, %f\n", P1g.x, P1g.y, P1g.z);
    // printf("Source Pressure: %f, %f\n", source_pressure.r, source_pressure.i);

    // This is the x offset from the base point to the approximate centriod of the pixel.
    float xoffset = delta * (xPnt - NumXpntsNegative) + delta / 2; // This value can be negative.
    // This is the y offset from the base point to the approximate centriod of the pixel.
    float yoffset = delta * yPnt + delta / 2;

    float3 xAxis = facets_xaxis[facet_num];
    float3 yAxis = facets_yaxis[facet_num];

    xAxis.x = xoffset * xAxis.x;
    xAxis.y = xoffset * xAxis.y;
    xAxis.z = xoffset * xAxis.z;

    yAxis.x = yoffset * yAxis.x;
    yAxis.y = yoffset * yAxis.y;
    yAxis.z = yoffset * yAxis.z;

    float3 facet_base = base_points[facet_num];
    float3 P2g;
    P2g.x = xAxis.x + yAxis.x + facet_base.x;
    P2g.y = xAxis.y + yAxis.y + facet_base.y;
    P2g.z = xAxis.z + yAxis.z + facet_base.z;

    // The distance from the source point to the facet point.
    float r_ij = sqrtf((P1g.x - P2g.x) * (P1g.x - P2g.x) + (P1g.y - P2g.y) * (P1g.y - P2g.y) + (P1g.z - P2g.z) * (P1g.z - P2g.z));

    // P2 = P2*exp(-i*k*r_sf)
    dcomplex i = devComplex(0, 1);
    dcomplex var = devCmul(i, k);
    var = devRCmul(r_ij, var);
    var = devCexp(var);                  // This has phase and attenuation.
    var = devCmul(var, source_pressure); // This includes the orginal pressure.
    // printf("Pressure prior to spreading at facet point: %f, %f\n", var.r, var.i);

    // Area1 = Pressure the 1Pa over 1m^2
    // Area2 = 4 * PI * r_sf * r_sf
    // atten_spread = Area1 / Area2 <--- important for other projections.

    // Point sources have pressure values @ RE 1 m
    // A_i = 4 * PI * 1^2
    // A_j = 4 * PI * r_sf * r_sf
    float A_r = 1 / (r_ij * r_ij);

    var = devRCmul(A_r, var);
    // printf("Spherical spread: %f\n", att_spread);

    if (devCabs(var) > 1.0)
    {
        printf("Source Point to Facet Error.\n");
        printf("Radius: %e\n", r_ij);
        printf("Spherical spread: %e\n", A_r);
        printf("Pressure add to field point prior to spreading: %e, %e\n", var.r, var.i);
        return;
    }

    // printf("Pressure at facet point: %f, %f\n", var.r, var.i);

    // Save the pressure to the facet pressure array.
    // Note var may be small and accumulate over may projects that why the complex numbers are doubles.
    atomicAddDouble(&(facets_Pressure[facet_num][yPnt * NumXpnts + xPnt].r), var.r);
    atomicAddDouble(&(facets_Pressure[facet_num][yPnt * NumXpnts + xPnt].i), var.i);
}

int CudaModelTes::ProjectSourcePointsToFacet()
{
    // Every facet can have a different number of pixels, where n = 1096^0.5 is the maximum number of pixels per facet.
    // printf("Host ProjectPointToFacet....\n");

    for (int source_point_num = 0; source_point_num < host_num_source_points; source_point_num++)
    {
        for (int object_num = 0; object_num < host_object_num_facets.size(); object_num++)
        {
            for (int facet_num = 0; facet_num < host_object_num_facets[object_num]; facet_num++)
            {
                int3 h_Facets_points = host_Object_Facets_points[object_num][facet_num];

                dim3 threadsPerBlock(h_Facets_points.x, h_Facets_points.y);
                dim3 numBlocks(1, 1);

                // printf("ThreadsPerBlock.x: %d, threadsPerBlock.y: %d\n", threadsPerBlock.x, threadsPerBlock.y);
                // printf("numBlocks.x: %d, numBlocks.y: %d\n", numBlocks.x, numBlocks.y);

                ProjectSourcePointToFacetKernel<<<numBlocks, threadsPerBlock>>>(
                    dev_k_wave,
                    dev_pixel_delta,
                    source_point_num,
                    facet_num,
                    dev_source_points_position,
                    dev_source_points_pressure,
                    dev_Object_Facets_points[object_num],
                    dev_Object_base_points[object_num],
                    dev_Object_Facets_xAxis[object_num],
                    dev_Object_Facets_yAxis[object_num],
                    dev_Object_Facets_PixelArea[object_num],
                    dev_Object_Facets_Pressure[object_num]);

                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                {
                    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
                    return 1;
                }
            }
        }
    }

    // More testing is required on large models to see how CUDA manages the cores.
    hipDeviceSynchronize();
    return 0;
}
