#include "hip/hip_runtime.h"
#include "FacetCuda.hpp"
#include "dcomplex.h"
#include "CudaUtils.cuh"

#include <thrust/complex.h>

#include <hip/hip_runtime.h>

__global__ void MakeSurface(double *Pr, double *Pi, hipSurfaceObject_t surface, int maxXpnt, float *stats)
{
    int xPnt = threadIdx.x;
    int yPnt = blockIdx.x;

    int index = yPnt * maxXpnt + xPnt;

    thrust::complex<float> p((float)Pr[index], (float)Pi[index]);

    float mag = abs(p) / stats[0];
    float phase = (atan2(p.imag(), p.real()) + M_PI) / (2.0f * M_PI);
    float2 value = make_float2(mag, phase);

    // value = (float)yPnt / (float)maxXpnt;
    surf2Dwrite(value, surface, xPnt * sizeof(float2), yPnt);

    // printf("Write surface: %f, %f, %f\n", value, stats[0], stats[1]);
}

void FacetCuda::WriteSurface(float *dev_frag_stats)
{
    MapToCuda();
    // Copy the data to the device.
    // Make the texture out of the real values.
    dim3 threadsPerBlock(numXpnts, 1);
    dim3 numBlocks(numYpnts, 1);
    MakeSurface<<<numBlocks, threadsPerBlock>>>(dev_Pr, dev_Pi, surface, numXpnts, dev_frag_stats);

    hipGraphicsUnmapResources(1, &cudaResource, 0);
    readyToRender = true;
}

__global__ void GetMaxValue(double *Pr, double *Pi, int maxXpnt, float *stats)
{
    int xPnt = threadIdx.x;
    int yPnt = blockIdx.x;

    int index = yPnt * maxXpnt + xPnt;

    thrust::complex<float> p((float)Pr[index], (float)Pi[index]);

    float mag = abs(p);

    atomicMaxFloat(&stats[0], mag);
    // atomicMinFloat(&stats[1], real);
    // atomicMaxFloat(&stats[2], abs);

    // printf("GetMaxValue: %e, %e, %e\n", stats[0], stats[1], stats[2]);
    //  printf("CurVal: %e, %e, %e\n", real, imag, abs);
}

void FacetCuda::GetSurfaceScalers(float *dev_frag_stats)
{
    dim3 threadsPerBlock(numXpnts, 1);
    dim3 numBlocks(numYpnts, 1);

    GetMaxValue<<<numBlocks, threadsPerBlock>>>(dev_Pr,
                                                dev_Pi,
                                                numXpnts,
                                                dev_frag_stats);
}