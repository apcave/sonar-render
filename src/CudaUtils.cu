#include "hip/hip_runtime.h"
#include "CudaUtils.cuh"

/**
 * @Brief Mutex style lock for double precision floating point numbers.
 *        This is a workaround for the lack of atomicAdd for double precision
 *       floating point numbers in CUDA on older devices.
 */
__device__ double atomicAddDouble(double *address, double val)
{
    unsigned long long int *address_as_ull = (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(__longlong_as_double(assumed) + val));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ float atomicMaxFloat(float *address, float val)
{
    int *address_as_int = (int *)address; // Treat the float as an int
    int old = *address_as_int, assumed;

    do
    {
        assumed = old;
        float old_val = __int_as_float(assumed);
        if (old_val >= val)
        {
            break; // No need to update if the current value is already greater or equal
        }
        old = atomicCAS(address_as_int, assumed, __float_as_int(val));
    } while (assumed != old);

    return __int_as_float(old);
}

__device__ float atomicMinFloat(float *address, float val)
{
    int *address_as_int = (int *)address; // Treat the float as an int
    int old = *address_as_int, assumed;

    do
    {
        assumed = old;
        float old_val = __int_as_float(assumed);
        if (old_val <= val)
        {
            break; // No need to update if the current value is already less or equal
        }
        old = atomicCAS(address_as_int, assumed, __float_as_int(val));
    } while (assumed != old);

    return __int_as_float(old);
}

__device__ float3 MakeVector(float3 Origin, float3 Dest)
{
    float3 Vc;
    Vc.x = Dest.x - Origin.x;
    Vc.y = Dest.y - Origin.y;
    Vc.z = Dest.z - Origin.z;
    return Vc;
}

__device__ float3 DivideVector(float3 vect, float variable)
{
    float3 dividedVect;
    dividedVect.x = vect.x / variable;
    dividedVect.y = vect.y / variable;
    dividedVect.z = vect.z / variable;
    return dividedVect;
}

__device__ float DotProduct(float3 v1, float3 v2)
{
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__device__ float GetVectorLength(float3 v1)
{
    return sqrtf(v1.x * v1.x + v1.y * v1.y + v1.z * v1.z);
}