#include "hip/hip_runtime.h"
#include "CudaModelTes.cuh"
#include "GeoMath.h"
#include "dcomplex.h"
#include "CudaUtils.cuh"

#include <stdio.h>

/**
 * @brief Kernel to waves from a point source to a facet.
 *
 * Note all pointers are to memory addresses on the device.
 * This function is not to be used with matrix compression.
 * The calculations are done in the global coordinate system.
 */
__global__ void ProjectSourcePointToFacetKernel(
    dcomplex *k_wave,
    float *pixel_delta,
    int source_point_num,
    int facet_num,
    float3 *source_points_position,
    dcomplex *source_points_pressure,
    int3 *facet_Points,
    float3 *base_points,
    float3 *facets_xaxis,
    float3 *facets_yaxis,
    float **facets_PixelArea,
    double *Pr_facet,
    double *Pi_facet)
{
    dcomplex k = *k_wave;
    float delta = *pixel_delta;

    // printf("k_wave: %f, %f\n", k.r, k.i);
    // printf("pixel_delta: %f\n", delta);

    // Kernel code to project point to point
    // printf("ThreadIdx.x: %d, ThreadIdx.y: %d, blockIdx.x: %d, blockDim.x: %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockDim.x);
    int xPnt = threadIdx.x;
    int yPnt = blockIdx.x;

    int NumXpnts = facet_Points[facet_num].x;
    // int NumYpnts = facet_Points[facet_num].y;
    int NumXpntsNegative = facet_Points[facet_num].z;

    int index_Bi = yPnt * NumXpnts + xPnt;

    float A_i = facets_PixelArea[facet_num][index_Bi];

    if (A_i == 0)
    {
        // printf("facets_PixelArea is zero\n");
        return;
    }

    float3 pg_i = source_points_position[source_point_num];
    dcomplex source_pressure = source_points_pressure[source_point_num];

    // This is the x offset from the base point to the approximate centriod of the pixel.
    float xoffset = delta * (xPnt - NumXpntsNegative) + delta / 2; // This value can be negative.
    // This is the y offset from the base point to the approximate centriod of the pixel.
    float yoffset = delta * yPnt + delta / 2;

    float3 xAxis = facets_xaxis[facet_num];
    float3 yAxis = facets_yaxis[facet_num];

    xAxis.x = xoffset * xAxis.x;
    xAxis.y = xoffset * xAxis.y;
    xAxis.z = xoffset * xAxis.z;

    yAxis.x = yoffset * yAxis.x;
    yAxis.y = yoffset * yAxis.y;
    yAxis.z = yoffset * yAxis.z;

    float3 facet_base = base_points[facet_num];
    float3 pg_j;
    pg_j.x = xAxis.x + yAxis.x + facet_base.x;
    pg_j.y = xAxis.y + yAxis.y + facet_base.y;
    pg_j.z = xAxis.z + yAxis.z + facet_base.z;

    // The distance from the source point to the facet point.
    float r_si = sqrtf((pg_i.x - pg_j.x) * (pg_i.x - pg_j.x) + (pg_i.y - pg_j.y) * (pg_i.y - pg_j.y) + (pg_i.z - pg_j.z) * (pg_i.z - pg_j.z));

    dcomplex i = devComplex(0, 1);
    dcomplex ik = devCmul(i, k);
    dcomplex exp_ikr = devRCmul(r_si, ik);
    exp_ikr = devCexp(exp_ikr); // This has phase and attenuation.

    double p = 1 / r_si;
    dcomplex G = devRCmul(p, exp_ikr);

    dcomplex R = devCmul(G, source_pressure); // Greens function times the source pressure.

    if (devCabs(G) > 1.0)
    {
        printf("Source Point to Facet Error.\n");
        printf("Radius: %e\n", r_si);
        printf("Spherical spread: %e\n", A_i);
        printf("Pressure add to field point prior to spreading: %e, %e\n", R.r, R.i);
        return;
    }

    atomicAddDouble(&Pr_facet[index_Bi], R.r);
    atomicAddDouble(&Pi_facet[index_Bi], R.i);
}

int CudaModelTes::ProjectSourcePointsToFacet()
{
    printf("Host ProjectPointToFacet....\n");

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Query device 0
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);

    printf("ProjectFromFacetsToFieldPoints .......\n");

    // Every facet can have a different number of pixels, where n = 1096^0.5 is the maximum number of pixels per facet.

    for (int source_point_num = 0; source_point_num < host_num_source_points; source_point_num++)
    {
        for (int object_num = 0; object_num < host_object_num_facets.size(); object_num++)
        {
            for (int facet_num = 0; facet_num < host_object_num_facets[object_num]; facet_num++)
            {
                int3 h_Facets_points = host_Object_Facets_points[object_num][facet_num];

                dim3 threadsPerBlock(h_Facets_points.x, 1);
                dim3 numBlocks(h_Facets_points.y, 1);

                // printf("Mutex Address : "
                //        "%p\n",
                //        mutex_in_cuda[object_num][facet_num]);

                // printf("Surface Real Address : "
                //        "%p\n",
                //        dev_Object_Facets_Surface_Pr[object_num][facet_num]);

                // printf("Surface Imaginary Address : "
                //        "%p\n",
                //        dev_Object_Facets_Surface_Pi[object_num][facet_num]);

                ProjectSourcePointToFacetKernel<<<numBlocks, threadsPerBlock>>>(
                    dev_k_wave,
                    dev_pixel_delta,
                    source_point_num,
                    facet_num,
                    dev_source_points_position,
                    dev_source_points_pressure,
                    dev_Object_Facets_points[object_num],
                    dev_Object_base_points[object_num],
                    dev_Object_Facets_xAxis[object_num],
                    dev_Object_Facets_yAxis[object_num],
                    dev_Object_Facets_PixelArea[object_num],
                    dev_object_facet_Pr[object_num][facet_num],
                    dev_object_facet_Pi[object_num][facet_num]);

                hipError_t err = hipGetLastError();
                if (err != hipSuccess)
                {
                    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
                    return 1;
                }
            }
        }
    }

    hipDeviceSynchronize();

    printf("ProjectSourcePointsToFacet done.\n");
    return 0;
}
